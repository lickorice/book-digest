
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void vecAddKernel(float* A, float* B, float* C, int n) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if(i < n) C[i] = A[i] + B[i];
}

// Compute vector sum d_C = d_A + d_B
void vecAdd(float* A, float* B, float* C, int n) {
    int size = n*sizeof(float);
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copy values in A, B, and C to device memory
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    // Call kernel function
    vecAddKernel<<<ceil(n/256.0), 256>>>(d_A, d_B, d_C, n);

    // Copy results back to C
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main(void) {
    int N = 1<<20;
    // Initialize float arrays:
    float *A, *B, *C, *expected;
    int size = N*sizeof(float);
    A = (float*) malloc(size);
    B = (float*) malloc(size);
    C = (float*) malloc(size);
    expected = (float*) malloc(size);

    for (int i = 0; i < N; ++i) {
        A[i] = i*5 + i;
        B[i] = i+20;
        expected[i] = A[i] + B[i];
    }

    vecAdd(A, B, C, N);

    int error = 0;
    for (int i = 0; i < N; ++i) {
        error += (expected[i] - C[i]);
    }

    std::cout << "Vector addition error: " << error << std::endl;
}